#include "cudatools.h"
#include "sgemm.h"
#include "matrix.h"
#include <chrono>
#include <iostream>

decltype(std::chrono::high_resolution_clock::now()) start;
decltype(std::chrono::high_resolution_clock::now()) stop;

inline void timing_start() {
	start = std::chrono::high_resolution_clock::now();
}

inline long timing_stop() {
	stop = std::chrono::high_resolution_clock::now();
	auto delta_t = std::chrono::duration_cast<std::chrono::microseconds>(stop - start); // fix type issue
	return delta_t.count();
}

void checkMatmul(Matrix & C, const Matrix & A, const Matrix & B) {
	C.download();
	uint N = C.N;
	std::cout << "Start errorcheck:" << std::endl;
	bool doBreak = false;
	for (uint i = 0; i < N; i++) {
		for (uint j = 0; j < N; j++) {
			if (Matrix::matmul(A, B, i, j) != C.get(i,j)) {
				std::cout << "Error detected: " << std::endl;
				std::cout << "i: " << i << " j: " << j << std::endl;
				doBreak = true;
				break;
			}
		}
		if (doBreak) break;
	}
	if (not doBreak) std::cout << "No errors" << std::endl;
}

constexpr long N = 2048; // 2048;
constexpr uint blocksize = 32; // length of one side!!!! -> blockDim = blocksize ** 2
static_assert(N%blocksize == 0);
int main () {
	Matrix A(N), B(N), C(N);
	// A.fill_random(0);
	A.fill_ones();
	A.upload();
	// B.fill_random(1);
	B.fill_ones();
	B.upload();
	C.fill_random(2);
	C.upload();


	dim3 gridDim(N/blocksize, N/blocksize, 1);
	dim3 blockDim(blocksize*blocksize, 1, 1);

	float alpha = 1;
	float beta = 0;
	unsigned reps = 50;
	std::cout << "STARTED TIMING" << std::endl;
	timing_start();
	for (unsigned rep = 0; rep < reps; rep++) {
		sgemm_coalesced<float, N, blocksize> <<< gridDim , blockDim >>> (C.d_data, A.d_data, B.d_data, alpha, beta);
		//sgemm_sharedmem<float, N, blocksize> <<< gridDim , blockDim >>> (C.d_data, A.d_data, B.d_data, alpha, beta);
		CLCE();
		CCE(hipDeviceSynchronize());
	}
	unsigned microsecs = timing_stop();
	std::cout << "BANDWIDTH (MByte/s): " << sizeof(float)*reps*(4*N*N)/(double)microsecs << std::endl;
	std::cout << "ARITHETICS (GFLOPS/s) (TODO): " << reps*(2*N*N*N)/((double)microsecs*1000) << std::endl;
	CLCE();


	// check for correctness
	C.download();
	uint i = 123;
	uint j = 456;
	std::cout << "i: " << i << " j: " << j << std::endl;
	std::cout << Matrix::matmul(A, B, i, j) << " <---> " << C.get(i,j) << std::endl;
	// checkMatrix(C, A, B);
}
